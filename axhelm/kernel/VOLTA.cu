#include "hip/hip_runtime.h"
/*

See LICENSE file.

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__forceinline__ __device__ __host__  int ijN(const int i, const int j, const int N){

  return i + j*N;

}

__forceinline__ __device__ __host__ int ijkN(const int i, const int j, const int k, const int N){

  return i + j*N + k*N*N;

}

__forceinline__ __device__ __host__ int ijklN(const int i, const int j, const int k, const int l, const int N){

  return i + j*N + k*N*N + l*N*N*N;

}

#define MAX_DOFS_1D 14
#define MAX_HALF_DOFS_1D 7

#define HALF_DOFS_1D ((p_Nq+1)/2)

#define NUM_DOFS_2D (p_Nq*p_Nq)
#define NUM_DOFS_3D (p_Nq*p_Nq*p_Nq)

#define p_Nblock 1

__constant__ dfloat const_DofToDofD[MAX_DOFS_1D*MAX_DOFS_1D];

__forceinline__ __device__ 
  void axhelmDevice(const int numElements,
		 const int element,
		 const dfloat lambda,
		 const dfloat * __restrict__ op,
		 const dfloat * __restrict__ DofToDofD,
		 dfloat * __restrict__ r_p,
		 dfloat * __restrict__ r_Ap){
  
  __shared__ dfloat s_p[p_Nblock][p_Nq][p_Nq];
  __shared__ dfloat s_Gpr[p_Nblock][p_Nq][p_Nq];
  __shared__ dfloat s_Gps[p_Nblock][p_Nq][p_Nq];
  
  // assumes NUM_DOFS_2D threads
  int t = threadIdx.x;
  int blk = threadIdx.y;
  
  int i = t%p_Nq;
  int j = t/p_Nq;
  
  for(int k = 0; k < p_Nq; k++) {
    r_Ap[k] = 0.f; // zero the accumulator
  }
  
  // Layer by layer
#pragma unroll
  for(int k = 0; k < p_Nq; k++) {

    // share r_p[k]
    __syncthreads();

    s_p[blk][j][i] = r_p[k];

    __syncthreads();
    
    dfloat G00 = 0, G01 =0, G02 =0, G11 =0, G12 =0, G22 =0, GWJ =0;
    
    // prefetch geometric factors
    const int gbase = element*p_Nggeo*NUM_DOFS_3D + ijkN(i,j,k,p_Nq);

    if(element<numElements){
      G00 = op[gbase+p_G00ID*NUM_DOFS_3D];
      G01 = op[gbase+p_G01ID*NUM_DOFS_3D];
      G02 = op[gbase+p_G02ID*NUM_DOFS_3D];
      G11 = op[gbase+p_G11ID*NUM_DOFS_3D];
      G12 = op[gbase+p_G12ID*NUM_DOFS_3D];
      G22 = op[gbase+p_G22ID*NUM_DOFS_3D];
      GWJ = op[gbase+p_GWJID*NUM_DOFS_3D];
    }
    
    dfloat pr = 0.f;
    dfloat ps = 0.f;
    dfloat pt = 0.f;

#pragma unroll
    for(int m = 0; m < p_Nq; m++) {
      int im = ijN(m,i,p_Nq);
      int jm = ijN(m,j,p_Nq);
      int km = ijN(m,k,p_Nq);
      pr += DofToDofD[im]*s_p[blk][j][m];
      ps += DofToDofD[jm]*s_p[blk][m][i];
      pt += DofToDofD[km]*r_p[m];
    }
    
    s_Gpr[blk][j][i] = (G00*pr + G01*ps + G02*pt);
    s_Gps[blk][j][i] = (G01*pr + G11*ps + G12*pt);
    
    dfloat Gpt = (G02*pr + G12*ps + G22*pt);
    
    dfloat Apk = GWJ*lambda*r_p[k];
    
    __syncthreads();
    
#pragma unroll
    for(int m = 0; m < p_Nq; m++){
      int mi = ijN(i,m,p_Nq);
      int mj = ijN(j,m,p_Nq);
      int km = ijN(m,k,p_Nq);
      Apk     += DofToDofD[mi]*s_Gpr[blk][j][m];
      Apk     += DofToDofD[mj]*s_Gps[blk][m][i];
      r_Ap[m] += DofToDofD[km]*Gpt; // DT(m,k)*ut(i,j,k,e)
    }
    
    r_Ap[k] += Apk;
  }
  
}

extern "C" __global__ void axhelm_v0(const int numElements,
		                  const dfloat * __restrict__ op,
		                  const dfloat * __restrict__ DofToDofD,
	       	                  const dfloat lambda,
		                  const dfloat * __restrict__ solIn,
		                  dfloat * __restrict__ solOut){
  
  __shared__ dfloat s_DofToDofD[NUM_DOFS_2D];

  dfloat r_q[p_Nq];
  dfloat r_Aq[p_Nq];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const int element = blockIdx.x*p_Nblock + blk;
  
  const unsigned int a = t%p_Nq;
  const unsigned int b = t/p_Nq;

  s_DofToDofD[t] = DofToDofD[t];
  
  if(element < numElements){
    for(int c=0;c<p_Nq;++c){
      
      int id = ijklN(a,b,c,element,p_Nq);
      
      r_q[c] = solIn[id];
    }
  }
  
  __syncthreads();
  
  axhelmDevice(numElements, element, lambda, op, s_DofToDofD, r_q, r_Aq);
  
  if(element<numElements){
#pragma unroll
    for(int c=0;c<p_Nq;++c){
      int id = ijklN(a,b,c,element,p_Nq);
      solOut[id] = r_Aq[c];
    }
  }
}
